#include <stdio.h>
#include <stdlib.h>
#include "hipsparse.h"
#include "hip/hip_runtime.h"
#include <getopt.h>

#include "gg.h"
#include "Timer.h"

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed with error (%d) at line %d\n",             \
                status, __LINE__);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

char *INPUT, *OUTPUT;
int QUIET = 0;
int CUDA_DEVICE = 0;
int DIS_NCOLORS = 0;

void usage(int argc, char *argv[]) {
    fprintf(stderr, "usage: %s [-q] [-g gpunum] [-o output-file] [-c output number of colors] graph-file\n", argv[0]);
}

void parse_args(int argc, char *argv[]) {
    char opts[] = "g:qo:c";
    int len = strlen(opts)+1;
    int c;

    while((c = getopt(argc, argv, opts)) != -1) {
        switch(c) {
            case 'q':
                QUIET = 1;
                break;
            case 'o':
                OUTPUT = optarg;
                break;
            case 'g':
                char *end;
                errno = 0;
                CUDA_DEVICE = strtol(optarg, &end, 10);
                if(errno != 0 || *end != '\0') {
                    fprintf(stderr, "Invalid GPU device '%s'. An integer must be specified.\n", optarg);
                    exit(EXIT_FAILURE);
                }
            case 'c':
                DIS_NCOLORS = 1;
                break;
            case '?':
                usage(argc, argv);
                exit(EXIT_FAILURE);
        }
    }

    if(optind < argc) {
        INPUT = argv[optind];
    }
    else {
        usage(argc, argv);
        exit(EXIT_FAILURE);
    }
}

void output(int m, int *coloring, int ncolors) {
    if(!QUIET) {
        if(DIS_NCOLORS) {
           printf("Chromatic number: %d", ncolors);
        }
        if(OUTPUT) {
            FILE *f;

            if(strcmp(OUTPUT, "-") == 0)
                f = stdout;
            else
                f = fopen(OUTPUT, "w");

            fprintf(f, "%d\n", m); // number of nodes
            for (int i = 0; i < m; i++)
                fprintf(f, "%d %d\n", i, coloring[i]);
        }
    }

}

int main(int argc, char *argv[]) {

    parse_args(argc, argv);

    // CSR matrix variables

    int m, nnz;
    float *val;

    // Load graph and set csr matrix variables

    CSRGraphTy g;
    g.read(INPUT);

    m = g.nnodes;
    nnz = g.nedges;
    val = (float*) malloc(nnz * sizeof(float));
    for(int i = 0; i < nnz; i++) val[i] = 1;

    int *d_rowPtr, *d_colInd;
    float *d_val;

    check_cuda(hipMalloc((void **)&d_rowPtr, (m + 1) * sizeof(int)));
    check_cuda(hipMalloc((void **)&d_colInd, nnz * sizeof(int)));
    check_cuda(hipMalloc((void **)&d_val, nnz * sizeof(int)));

    check_cuda(hipMemcpy(d_rowPtr, g.row_start, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
    check_cuda(hipMemcpy(d_colInd, g.edge_dst, nnz * sizeof(int), hipMemcpyHostToDevice));
    check_cuda(hipMemcpy(d_val, val, nnz * sizeof(int), hipMemcpyHostToDevice));

    // Allocate memory for storing coloring info

    int ncolors = 0, *coloring, *reordering;
    float fraction = 1.0;
    int *d_coloring, *d_reordering;

    coloring = (int *) malloc(m * sizeof(int));
    reordering = (int *) malloc(m * sizeof(int));

    check_cuda(hipMalloc((void **)&d_coloring, m * sizeof(int))); 
    check_cuda(hipMalloc((void **)&d_reordering, m * sizeof(int))); 

    // Create handle for cuSPARSE context 

    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Create Matrix descriptor and coloring info

    hipsparseMatDescr_t descr;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr));

    hipsparseColorInfo_t info;
    CHECK_CUSPARSE(hipsparseCreateColorInfo(&info));

    // Invoking csrcolor (and timing it)

    ggc::Timer k("cusparseScsrsolor");
    k.start();
    CHECK_CUSPARSE(hipsparseScsrcolor(handle, m, nnz, descr, d_val, d_rowPtr, d_colInd, &fraction, &ncolors, d_coloring, d_reordering, info));
    hipDeviceSynchronize();
    k.stop();

    // Copy coloring back to host

    check_cuda(hipMemcpy(coloring, d_coloring, m * sizeof(int), hipMemcpyDeviceToHost));
    check_cuda(hipMemcpy(reordering, d_reordering, m * sizeof(int), hipMemcpyDeviceToHost));

    // Output

    fprintf(stderr, "Total time: %llu ms\n", k.duration_ms());
    fprintf(stderr, "Total time: %llu ns\n", k.duration());

    output(m, coloring, ncolors);

    // Release memory acquired by descr and info

    CHECK_CUSPARSE(hipsparseDestroyColorInfo(info));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));

    // Release CPU side res used by cuSPARSE

    CHECK_CUSPARSE(hipsparseDestroy(handle));

    // Free allocated memory

    check_cuda(hipFree(d_reordering));
    check_cuda(hipFree(d_coloring));
    check_cuda(hipFree(d_val));
    check_cuda(hipFree(d_colInd));
    check_cuda(hipFree(d_rowPtr));

    free(reordering);
    free(coloring);
    free(val);

    return 0;
}
